#include <hip/hip_runtime.h>

#include <iostream>
#include <cmath>

using namespace std;

__device__ int dev_N;

__global__ void gemm_baseline(float *A, float *B, float *C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < dev_N && j < dev_N )
    {
        C[j * dev_N  + i] = 0;
        for (int k = 0; k < dev_N ; k++)
            C[j * dev_N  + i] += A[j * dev_N  + k] * B[k * dev_N  + i];
    }
}

void gemm_verify(float *A, float *B, float *C, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            float d = 0;
            for (int k = 0; k < N; k++)
            {
                d += A[i * N + k] * B[k * N + j];
            }
            if (C[i * N + j] - d > 1e-3 || C[i * N + j] - d < -1e-3)
            {
                printf("Error: C[%d] = %f, C_baseline[%d] = %f\n", i * N + j, C[i], i * N + j, d);
                return;
            }
        }
    }
}

int main(int argc, char const *argv[])
{
    int N = (1 << atoi(argv[1]));
    int BLOCK_SIZE = (1 << atoi(argv[2]));

    hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));

    // malloc
    float *A = (float *)malloc(N * N * sizeof(float));
    float *B = (float *)malloc(N * N * sizeof(float));
    float *C = (float *)malloc(N * N * sizeof(float));

    // random initialize A, B
    srand((unsigned int)time(0));
    for (int i = 0; i < N * N; i++)
    {
        A[i] = rand() / (float)RAND_MAX;
        B[i] = rand() / (float)RAND_MAX;
    }

    // cumalloc
    float *A_device, *B_device, *C_device;
    hipMalloc((void **)&A_device, N * N * sizeof(float));
    hipMalloc((void **)&B_device, N * N * sizeof(float));
    hipMalloc((void **)&C_device, N * N * sizeof(float));
    hipMemcpy(A_device, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_device, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // define gridsize & blocksize
    dim3 grid(N / BLOCK_SIZE, N / BLOCK_SIZE, 1), block(BLOCK_SIZE, BLOCK_SIZE, 1);

    // timing
    hipEvent_t start, stop;

    // create CUDA events for timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // record start event
    hipEventRecord(start, 0);

    // launch kernel
    gemm_baseline<<<grid, block>>>(A_device, B_device, C_device);

    // record stop event
    hipEventRecord(stop, 0);

    // synchronize events
    hipEventSynchronize(stop);

    // calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // print elapsed time
    printf("Time: %.3f ms\n", elapsedTime);

    hipMemcpy(C, C_device, N * N * sizeof(float), hipMemcpyDeviceToHost);
    gemm_verify(A, B, C, N);
    

    hipFree(A_device);
    hipFree(B_device);
    hipFree(C_device);
    free(A);
    free(B);
    free(C);
}