#include <hip/hip_runtime.h>

#include <iostream>
#include <cmath>
#define BLOCK_SIZE 16

using namespace std;

__device__ int dev_N;

__global__ void gemm_baseline(float *A, float *B, float *C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < dev_N && j < dev_N)
    {
        C[j * dev_N + i] = 0;
        int b_idx = blockIdx.x * blockDim.x;
        for (int a_idx = blockIdx.y * blockDim.y * dev_N;
             a_idx < blockIdx.y * blockDim.y * dev_N + dev_N - 1; a_idx += blockDim.x, b_idx += blockDim.y * dev_N)
        {
            __shared__ float Sub_A[BLOCK_SIZE][BLOCK_SIZE];
            __shared__ float Sub_B[BLOCK_SIZE][BLOCK_SIZE];

            Sub_A[threadIdx.y][threadIdx.x] = A[a_idx + threadIdx.y * dev_N + threadIdx.x];
            Sub_B[threadIdx.y][threadIdx.x] = B[b_idx + threadIdx.y * dev_N + threadIdx.x];

            __syncthreads();

            for (int k = 0; k < BLOCK_SIZE; k++)
            {
                C[j * dev_N + i] += Sub_A[threadIdx.y][k] * Sub_B[k][threadIdx.x];
            }

            __syncthreads();
        }
    }
}

void gemm_verify(float *A, float *B, float *C, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            float d = 0;
            for (int k = 0; k < N; k++)
            {
                d += A[i * N + k] * B[k * N + j];
            }
            if (C[i * N + j] - d > 1e-3 || C[i * N + j] - d < -1e-3)
            {
                printf("Error: C[%d] = %f, C_baseline[%d] = %f\n", i * N + j, C[i], i * N + j, d);
                return;
            }
        }
    }
}

int main(int argc, char const *argv[])
{
    int N = (1 << atoi(argv[1]));

    hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));

    // malloc
    float *A = (float *)malloc(N * N * sizeof(float));
    float *B = (float *)malloc(N * N * sizeof(float));
    float *C = (float *)malloc(N * N * sizeof(float));

    // random initialize A, B
    srand((unsigned int)time(0));
    for (int i = 0; i < N * N; i++)
    {
        A[i] = rand() / (float)RAND_MAX;
        B[i] = rand() / (float)RAND_MAX;
    }

    // cumalloc
    float *A_device, *B_device, *C_device;
    hipMalloc((void **)&A_device, N * N * sizeof(float));
    hipMalloc((void **)&B_device, N * N * sizeof(float));
    hipMalloc((void **)&C_device, N * N * sizeof(float));
    hipMemcpy(A_device, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_device, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // define gridsize & blocksize
    dim3 grid(N / BLOCK_SIZE, N / BLOCK_SIZE, 1), block(BLOCK_SIZE, BLOCK_SIZE, 1);

    // timing
    hipEvent_t start, stop;

    // create CUDA events for timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // record start event
    hipEventRecord(start, 0);

    // launch kernel
    gemm_baseline<<<grid, block>>>(A_device, B_device, C_device);

    // record stop event
    hipEventRecord(stop, 0);

    // synchronize events
    hipEventSynchronize(stop);

    // calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // print elapsed time
    printf("Time: %.3f ms\n", elapsedTime);

    hipMemcpy(C, C_device, N * N * sizeof(float), hipMemcpyDeviceToHost);
    gemm_verify(A, B, C, N);

    hipFree(A_device);
    hipFree(B_device);
    hipFree(C_device);
    free(A);
    free(B);
    free(C);
}